#include "hip/hip_runtime.h"
#include "descend.cuh"
auto vlc_encode(const descend::array<descend::u8, 4> *const h_source_data,
                const descend::u32 *const h_codewords,
                const descend::u32 *const h_codewordlens,
                descend::u32 *const h_out_data, descend::u32 *const h_out_idx)
-> void {
    {
        auto gpu = descend::gpu_device(0);
        const auto source_data = descend::gpu_alloc_copy<
                descend::array<descend::array<descend::u8, 4>, (4 * 256)>>(
                (&gpu), (&(*h_source_data)));
        const auto codewords =
                descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
                        (&gpu), (&(*h_codewords)));
        const auto codewordlens =
                descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
                        (&gpu), (&(*h_codewordlens)));
        auto out_data =
                descend::gpu_alloc_copy<descend::array<descend::u32, (4 * 256)>>(
                        (&gpu), (&(*h_out_data)));
        auto out_idx = descend::gpu_alloc_copy<descend::array<descend::u32, 4>>(
                (&gpu), (&(*h_out_idx)));
        descend::exec<4, 256>(
                (&gpu),
                [] __device__(const descend::array<descend::u8, 4> *const p0,
                              const descend::u32 *const p1,
                              const descend::u32 *const p2, descend::u32 *const p3,
                              descend::u32 *const p4, std::size_t d,
                              std::size_t tmp_d_item_i) -> void {
                    {

                        __shared__ descend::u32 sm_cw[256];
                        __shared__ descend::u32 sm_cwl[256];
                        __shared__ descend::u32 sm_as[256];
                        __shared__ descend::u32 sm_kcmax[1];
                        {

                            descend::u64 codeword;
                            descend::u32 codewordlen;
                            descend::u32 kc;
                            descend::u32 startbit;

                            {
                                sm_cw[threadIdx.x] = p1[threadIdx.x];
                                sm_cwl[threadIdx.x] = p2[threadIdx.x];
                            }
                            {
                                const auto foo = sm_as;

                                {
                                    codeword = 0;
                                    codewordlen = 0;
                                    const auto tmp_d_item =
                                            p0[((blockIdx.x * 256) + threadIdx.x)];
                                    descend::u8 tmp_d_item_i;
                                    tmp_d_item_i = tmp_d_item[0];
                                    auto tmpcw = sm_cw[tmp_d_item_i];
                                    auto tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64)(tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = tmp_d_item[1];
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64)(tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = tmp_d_item[2];
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64)(tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = tmp_d_item[3];
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64)(tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    (&(*foo))[threadIdx.x] = codewordlen;
                                }
                            }
                            {
                                const auto foo = sm_as;
                                for (std::size_t d = 128; (d > 0); d = (d / 2)) {

                                    if ((threadIdx.x < d)) {
                                        (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((256 / d) - 1))] =
                                                ((&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                            ((256 / d) - 1))] +
                                                 (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                            ((128 / d) - 1))]);
                                    } else {
                                    }
                                    __syncthreads();
                                }
                            }

                            if ((threadIdx.x < 1)) {
                                sm_as[((threadIdx.x - 0) + (256 - 1))] = 0;
                            } else {
                            }
                            __syncthreads();
                            {
                                const auto foo = sm_as;
                                for (std::size_t d = 1; (d <= 128); d = (d * 2)) {

                                    if ((threadIdx.x < d)) {
                                        const auto t = (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                                  ((128 / d) - 1))];
                                        (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((128 / d) - 1))] =
                                                (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                           ((256 / d) - 1))];
                                        (&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((256 / d) - 1))] =
                                                ((&(*foo))[(((threadIdx.x - 0) * (256 / d)) +
                                                            ((256 / d) - 1))] +
                                                 t);
                                    } else {
                                    }
                                    __syncthreads();
                                }
                            }
                            if ((threadIdx.x < 255)) {

                            } else {

                                {
                                    p4[((blockIdx.x * 1) + (threadIdx.x - 255))] =
                                            (sm_as[255] + codewordlen);
                                    sm_kcmax[(threadIdx.x - 255)] =
                                            ((sm_as[255] + codewordlen) / 32);
                                }
                            }
                            __syncthreads();
                            {
                                const auto foo = sm_as;

                                {
                                    kc = ((&(*foo))[threadIdx.x] / 32);
                                    startbit = ((&(*foo))[threadIdx.x] % 32);
                                    (&(*foo))[threadIdx.x] = 0;
                                }
                            }

                            {
                                descend::u32 wrbits;
                                if ((codewordlen > (32 - startbit))) {
                                    wrbits = (32 - startbit);
                                } else {
                                    wrbits = codewordlen;
                                }
                                auto tmpcw =
                                        (descend::u32)((codeword >> (codewordlen - wrbits)));
                                codewordlen = (codewordlen - wrbits);
                                if ((codewordlen > 0)) {
                                    if ((codewordlen > 32)) {
                                        wrbits = 32;
                                    } else {
                                        wrbits = codewordlen;
                                    }
                                    codewordlen = (codewordlen - wrbits);
                                    descend::u32 tmp;
                                    tmp = ((1 << wrbits) - 1);
                                    tmpcw = (descend::u32)((codeword >> codewordlen));
                                    tmpcw = (tmpcw & tmp);
                                }
                                if ((codewordlen > 0)) {
                                    descend::u64 tmp;
                                    tmp = ((1 << codewordlen) - 1);
                                    tmpcw = (descend::u32)((codeword & tmp));
                                }
                            }

                            { p3[((blockIdx.x * 256) + threadIdx.x)] = sm_as[threadIdx.x]; }
                        }
                    }
                },
                (&source_data), (&codewords), (&codewordlens), (&out_data), (&out_idx),
                d, tmp_d_item_i);
        descend::copy_to_host<descend::array<descend::u32, (4 * 256)>>((&out_data),
                                                                       h_out_data);
        descend::copy_to_host<descend::array<descend::u32, 4>>((&out_idx),
                                                               h_out_idx);
    }
}