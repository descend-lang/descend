#include "hip/hip_runtime.h"
#include <iostream>

#define BENCH
#include "descend.cuh"
template<std::size_t m>
auto scan_inplace(
        descend::i32 * const data
) -> void {
    auto accum = 0;
    for (std::size_t _i__0 = 0; _i__0 < m; _i__0 = _i__0 + 1) {
        const auto next = data[_i__0] + accum;
        data[_i__0] = accum;
        accum = next;
    }
}

template<std::size_t n, std::size_t gridDim>
auto scan(
        const descend::i32 * const ha_array,
        descend::i32 * const h_output,
        descend::i32 * const h_block_sums
) -> void {
    const auto gpu = descend::gpu_device(0);
    const auto a_array = descend::gpu_alloc<descend::array<descend::i32, n>>(&gpu, ha_array);
    auto out_array = descend::gpu_alloc<descend::array<descend::i32, n>>(&gpu, &*h_output);
    auto block_sums = descend::gpu_alloc<descend::array<descend::i32, gridDim>>(&gpu, &*h_block_sums);
    descend::exec<gridDim, 32>(&gpu, [] __device__ (
            const descend::i32 * const p0,
            descend::i32 * const p1,
            descend::i32 * const p2) -> void {
        __shared__ descend::i32 tmp[64];
        tmp[threadIdx.x] = p0[blockIdx.x * 64 + threadIdx.x];
        tmp[threadIdx.x + 32] = p0[blockIdx.x * 64 + threadIdx.x + 32];
        __syncthreads();
        for (descend::i32 d = 32; d > 0; d = d / 2) {
            if (threadIdx.x < d)
            {
                tmp[threadIdx.x * 64 / d + 64 / d - 1] = tmp[threadIdx.x * 64 / d + 64 / d - 1] + tmp[threadIdx.x * 64 / d + 32 / d - 1];
            }
            __syncthreads();
        }
        if (threadIdx.x < 1)
        {
            p2[blockIdx.x * 1 + threadIdx.x] = tmp[threadIdx.x + 63];
            tmp[threadIdx.x + 63] = 0;
        }
        __syncthreads();
        for (descend::i32 d = 1; d <= 32; d = d * 2) {
            if (threadIdx.x < d)
            {
                const auto t = tmp[threadIdx.x * 64 / d + 32 / d - 1];
                tmp[threadIdx.x * 64 / d + 32 / d - 1] = tmp[threadIdx.x * 64 / d + 64 / d - 1];
                tmp[threadIdx.x * 64 / d + 64 / d - 1] = tmp[threadIdx.x * 64 / d + 64 / d - 1] + t;
            }
            __syncthreads();
        }
        p1[blockIdx.x * 64 + threadIdx.x] = tmp[threadIdx.x];
        p1[blockIdx.x * 64 + threadIdx.x + 32] = tmp[threadIdx.x + 32];
        __syncthreads();

    }, &a_array, &out_array, &block_sums);
    descend::copy_to_host<descend::array<descend::i32, gridDim>>(&block_sums, &*h_block_sums);
    scan_inplace<gridDim>(&*h_block_sums);
    descend::copy_to_gpu<descend::array<descend::i32, gridDim>>(&block_sums, &*h_block_sums);
    descend::exec<gridDim, 64>(&gpu, [] __device__ (
            descend::i32 * const p0,
            const descend::i32 * const p1) -> void {
        p0[blockIdx.x * 64 + threadIdx.x] = p0[blockIdx.x * 64 + threadIdx.x] + p1[blockIdx.x];
        __syncthreads();

    }, &out_array, &block_sums);
    descend::copy_to_host<descend::array<descend::i32, n>>(&out_array, &*h_output);
}

//
// Manually added, not generated
//
descend::Benchmark benchmark{descend::BenchConfig({"scan", "add"})};
auto main() -> int {
    const auto NUM_RUNS = 10;
    const auto BASE_SIZE = 1024;
    const auto BLOCK_DIM = 32;
    const auto N = BASE_SIZE*BLOCK_DIM*2;
    const auto GRID_DIM = BASE_SIZE;
    // Check results
    auto gold = descend::HeapBuffer<descend::array<descend::i32, N>>(3);
    scan_inplace<N>(&gold);

    for (size_t i = 0; i < NUM_RUNS; i++) {
        // Run scan
        const auto ha_array = descend::HeapBuffer<descend::array<descend::i32, N>>(3);
        auto h_output = descend::HeapBuffer<descend::array<descend::i32, N>>(0);
        auto h_block_sums = descend::HeapBuffer<descend::array<descend::i32, N / 64>>(0);
        scan<N, GRID_DIM>(&ha_array, &h_output, &h_block_sums);

        for (int i = 0; i < N; i++) {
            if (h_output[i] != gold[i]) {
                std::cout << "Error at " << i << ": Expected `" << gold[i]
                          << "` but found `" << h_output[i] << "`" << std::endl;
                std::cout << "Next 10 lines:" << std::endl;
                for (int j = i; j < i + 10; j++)
                    std::cout << "Expected: " << gold[j] << " Found: " << h_output[i] << std::endl;
                exit(EXIT_FAILURE);
            }
        }
    }

    std::cout << benchmark.avg_to_csv();
};
