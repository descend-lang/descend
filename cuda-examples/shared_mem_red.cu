#include "hip/hip_runtime.h"
#include "descend.cuh"

template<std::size_t n>
auto reduce_shared_mem(
        const descend::i32 * const ha_array,
        descend::i32 * const h_output
) -> void {
    const auto gpu = descend::gpu_device(0);
    const auto a_array = descend::gpu_alloc<descend::array<descend::i32, n>>(&gpu, ha_array);
    auto out_array = descend::gpu_alloc<descend::array<descend::i32, 64>>(&gpu, &*h_output);
    descend::exec<64, 1024>(&gpu, [] __device__ (
            const descend::i32 * const p0,
            descend::i32 * const p1) -> void {
        __shared__ descend::i32 tmp[1024];
        tmp[threadIdx.x] = p0[blockIdx.x * 1024 + threadIdx.x];
        __syncthreads();
        for (descend::i32 k = 512; k > 0; k = k / 2) {
            if (threadIdx.x < k)
            {
                tmp[threadIdx.x] = tmp[threadIdx.x] + tmp[threadIdx.x + k];
            }

            __syncthreads();
        }

        if (threadIdx.x < 1)
        {
            p1[blockIdx.x * 1 + threadIdx.x] = tmp[threadIdx.x];
        }

        __syncthreads();
        ;
    }, &a_array, &out_array);
    descend::copy_to_host<descend::array<descend::i32, 64>>(&out_array, h_output);
}

auto main() -> int {
    const auto ha_array = descend::HeapBuffer<descend::array<descend::i32, 64*1024>>(1);
    auto h_output = descend::HeapBuffer<descend::array<descend::i32, 64>>(0);
    reduce_shared_mem<64*1024>(&ha_array, &h_output);

    for (size_t i = 0; i < 64; i++) {
        if (h_output[i] != 1024) {
            std::cout << "At i = " << i << "Wrong number. Found " << h_output[i] << " instead of 1024.";
            exit(EXIT_FAILURE);
        }
    }
    exit(EXIT_SUCCESS);
}
