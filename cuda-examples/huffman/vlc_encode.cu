#include "hip/hip_runtime.h"
// =====================================================================================================================
//   code generated from ../examples/infer/huffman/vlc_encode.desc
// =====================================================================================================================

#include "descend.cuh"
/*
function declarations
*/
template <std::size_t gs>
__host__ auto vlc_encode(const descend::u32 *const h_source_data,
                         const descend::u32 *const h_codewords,
                         const descend::u32 *const h_codewordlens,
                         descend::u32 *const h_out,
                         descend::u32 *const h_out_idx) -> void;
template <std::size_t gs>
__global__ auto gpu_vlc_encode(const descend::u32 *const g_source_data,
                               const descend::u32 *const g_codewords,
                               const descend::u32 *const g_codewordlens,
                               descend::u32 *const g_out,
                               descend::u32 *const g_out_idx) -> void;
/*
function definitions
*/
template <std::size_t gs>
__host__ auto vlc_encode(const descend::u32 *const h_source_data,
                         const descend::u32 *const h_codewords,
                         const descend::u32 *const h_codewordlens,
                         descend::u32 *const h_out,
                         descend::u32 *const h_out_idx) -> void {
    auto gpu = descend::gpu_device(0);
    const auto g_source_data =
    descend::gpu_alloc_copy<descend::array<descend::u32, (gs * 256)>>(
            (&gpu), (&(*h_source_data)));
    const auto g_codewords =
    descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
            (&gpu), (&(*h_codewords)));
    const auto g_codewordlens =
    descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
            (&gpu), (&(*h_codewordlens)));
    auto g_out =
    descend::gpu_alloc_copy<descend::array<descend::u32, (gs * 256)>>(
            (&gpu), (&(*h_out)));
    auto g_out_idx = descend::gpu_alloc_copy<descend::array<descend::u32, gs>>(
            (&gpu), (&(*h_out_idx)));
    gpu_vlc_encode<gs>
    <<<dim3(gs, 1, 1), dim3(256, 1, 1),
    (((((0 + (4 * (1 * 256))) + (4 * (1 * 256))) + (4 * (1 * 256))) +
      (4 * (1 * 256))) +
     (4 * (1 * 1)))>>>((&g_source_data), (&g_codewords), (&g_codewordlens),
                       (&g_out), (&g_out_idx));
    descend::copy_to_host<descend::array<descend::u32, (gs * 256)>>((&g_out),
            h_out);
    descend::copy_to_host<descend::array<descend::u32, gs>>((&g_out_idx),
            h_out_idx);
}

template <std::size_t gs>
__global__ auto gpu_vlc_encode(const descend::u32 *const g_source_data,
                               const descend::u32 *const g_codewords,
                               const descend::u32 *const g_codewordlens,
                               descend::u32 *const g_out,
                               descend::u32 *const g_out_idx) -> void {
    extern __shared__ descend::byte $buffer[];
    descend::u32 *const s_codewords = (descend::u32 *)((&$buffer[0]));
    descend::u32 *const s_codewordlens =
            (descend::u32 *)((&s_codewords[(1 * 256)]));
    descend::u32 *const s_result_locations =
            (descend::u32 *)((&s_codewordlens[(1 * 256)]));
    descend::u32 *const s_block_out =
            (descend::u32 *)((&s_result_locations[(1 * 256)]));
    descend::u32 *const s_last_index_to_copy =
            (descend::u32 *)((&s_block_out[(1 * 256)]));

    {

        auto l_thread_out = ((descend::u64)0);
        auto l_thread_out_len = 0u;
        auto l_thread_start_value = 0u;
        auto l_thread_start_bit = 0u;
        {

            {
                const auto g_codeword_item = (&g_codewords[(threadIdx.x - 0)]);
                const auto g_codewordlen_item = (&g_codewordlens[(threadIdx.x - 0)]);
                const auto s_codeword_item = (&(&(*s_codewords))[(threadIdx.x - 0)]);
                const auto s_codewordlen_item =
                        (&(&(*s_codewordlens))[(threadIdx.x - 0)]);
                (*s_codeword_item) = (*g_codeword_item);
                (*s_codewordlen_item) = (*g_codewordlen_item);
            }
        }

        __syncthreads();
        {
            {
                const auto g_source_data_item =
                        (&g_source_data[(((blockIdx.x - 0) * 256) + (threadIdx.x - 0))]);
                const auto s_result_location_item =
                        (&(&(*s_result_locations))[(threadIdx.x - 0)]);
                const auto tmp_source_data_item = (*g_source_data_item);
                auto tmp_source_data_item_i = ((descend::u8)0);
                auto tmp_cw = 0u;
                auto tmp_cwl = 0u;
                for (std::size_t i = 0; (i < 4); i = (i + 1u)) {
                    tmp_source_data_item_i = (descend::u8)(
                            (tmp_source_data_item >> (8 * (3 - (descend::i32)(i)))));
                    tmp_cw = s_codewords[tmp_source_data_item_i];
                    tmp_cwl = s_codewordlens[tmp_source_data_item_i];
                    l_thread_out = ((l_thread_out << tmp_cwl) | (descend::u64)(tmp_cw));
                    l_thread_out_len = (l_thread_out_len + tmp_cwl);
                }

                (*s_result_location_item) = l_thread_out_len;
            }
        }

        __syncthreads();
        {
            auto s_result_locations_ref = (&(*s_result_locations));
            for (std::size_t d = 128; (d > 0u); d = (d / 2u)) {
                if (((threadIdx.x - 0) < d)) {
                    {
                        (&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                      ((256 / d) - 1))] =
                                ((&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                               ((256 / d) - 1))] +
                                 (&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                               ((128 / d) - 1))]);
                    }
                } else {
                }

                __syncthreads();
            }
        }

        {

            if (((threadIdx.x - 0) < 1)) {
                {
                    const auto last =
                            (&(&(*s_result_locations))[((threadIdx.x - 0) + (256 - 1))]);
                    (*last) = 0u;
                }
            } else {
            }
        }

        __syncthreads();
        {
            auto s_result_locations_ref = (&(*s_result_locations));
            for (std::size_t d = 1; (d <= 128); d = (d * 2u)) {
                if (((threadIdx.x - 0) < d)) {
                    {
                        const auto t = (&(*s_result_locations_ref))[(
                                ((threadIdx.x - 0) * (256 / d)) + ((128 / d) - 1))];
                        (&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                      ((128 / d) - 1))] =
                                (&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                              ((256 / d) - 1))];
                        (&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                      ((256 / d) - 1))] =
                                ((&(*s_result_locations_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                               ((256 / d) - 1))] +
                                 t);
                    }
                } else {
                }

                __syncthreads();
            }
        }

        {
            if (((threadIdx.x - 0) < 255)) {
            } else {
                {
                    const auto g_out_idx_block_item =
                            (&g_out_idx[(((blockIdx.x - 0) * 1) + (threadIdx.x - 255))]);
                    const auto s_last_index_to_copy_item =
                            (&(&(*s_last_index_to_copy))[(threadIdx.x - 255)]);
                    (*g_out_idx_block_item) =
                            (s_result_locations[255] + l_thread_out_len);
                    (*s_last_index_to_copy_item) =
                            ((s_result_locations[255] + l_thread_out_len) / 32u);
                }
            }
        }

        __syncthreads();
        {
            {
                const auto s_result_locations_item =
                        (&(&(*s_result_locations))[(threadIdx.x - 0)]);
                const auto s_block_out_item = (&(&(*s_block_out))[(threadIdx.x - 0)]);
                l_thread_start_value = ((*s_result_locations_item) / 32u);
                l_thread_start_bit = ((*s_result_locations_item) % 32u);
                descend::atomic_store(
                        descend::atomic_ref<descend::u32>((*s_block_out_item)), 0u);
            }
        }

        __syncthreads();
        {
            const auto g_out_item =
                    (&g_out[(((blockIdx.x - 0) * 256) + (threadIdx.x - 0))]);
            const auto s_block_out_item = (&(&(*s_block_out))[(threadIdx.x - 0)]);
            descend::u32 wrbits;
            if ((l_thread_out_len > (32u - l_thread_start_bit))) {
                wrbits = (32u - l_thread_start_bit);
            } else {
                wrbits = l_thread_out_len;
            }

            auto tmpcw =
                    (descend::u32)((l_thread_out >> (l_thread_out_len - wrbits)));
            descend::atomic_fetch_or(
                    descend::atomic_ref<descend::u32>(s_block_out[l_thread_start_value]),
                    (tmpcw << ((32u - l_thread_start_bit) - wrbits)));
            l_thread_out_len = (l_thread_out_len - wrbits);
            if ((l_thread_out_len > 0u)) {
                if ((l_thread_out_len > 32u)) {
                    wrbits = 32u;
                } else {
                    wrbits = l_thread_out_len;
                }

                l_thread_out_len = (l_thread_out_len - wrbits);
                tmpcw = ((descend::u32)((l_thread_out >> l_thread_out_len)) &
                         ((1u << wrbits) - 1u));
                descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                 s_block_out[(l_thread_start_value + 1)]),
                                         (tmpcw << (32u - wrbits)));
            }

            if ((l_thread_out_len > 0u)) {
                tmpcw = (descend::u32)(
                        (l_thread_out &
                         ((((descend::u64)1) << l_thread_out_len) - ((descend::u64)1))));
                descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                 s_block_out[(l_thread_start_value + 2)]),
                                         (tmpcw << (32u - l_thread_out_len)));
            }

            __syncthreads();
            if ((descend::thread_id_x() <= s_last_index_to_copy[0])) {
                (*g_out_item) = descend::atomic_load(
                        descend::atomic_ref<descend::u32>((*s_block_out_item)));
            }
        }
    }
}