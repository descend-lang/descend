#include "hip/hip_runtime.h"
// code generated from ../examples/infer/huffman/vlc_encode.desc with a hand written shfl scan
// 2 __syncthreads operations have been inserted manually

#include "../descend.cuh"

auto vlc_encode(const descend::u32 *const h_source_data,
                const descend::u32 *const h_codewords,
                const descend::u32 *const h_codewordlens,
                descend::u32 *const h_out_data, descend::u32 *const h_out_idx)
-> void {
    {
        descend::Gpu gpu = descend::gpu_device(0);
        const const GpuBuffer<descend::array<descend::u32, (64 * 256)>>
                source_data =
                descend::gpu_alloc_copy<descend::array<descend::u32, (64 * 256)>>(
                        (&gpu), (&(*h_source_data)));
        const const GpuBuffer<descend::array<descend::u32, 256>> codewords =
                descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
                        (&gpu), (&(*h_codewords)));
        const const GpuBuffer<descend::array<descend::u32, 256>> codewordlens =
                descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
                        (&gpu), (&(*h_codewordlens)));
        GpuBuffer<descend::array<descend::u32, (64 * 256)>> out_data =
                descend::gpu_alloc_copy<descend::array<descend::u32, (64 * 256)>>(
                        (&gpu), (&(*h_out_data)));
        GpuBuffer<descend::array<descend::u32, 64>> out_idx =
                descend::gpu_alloc_copy<descend::array<descend::u32, 64>>(
                        (&gpu), (&(*h_out_idx)));
        descend::exec<64, 256>(
                (&gpu),
                [] __device__(const descend::u32 *const p0,
                              const descend::u32 *const p1,
                              const descend::u32 *const p2, descend::u32 *const p3,
                              descend::u32 *const p4) -> void {
                    {

                        __shared__ descend::u32 sm_cw[256];
                        __shared__ descend::u32 sm_cwl[256];
                        __shared__ descend::u32 sm_scan_arr[7];
                        __shared__ descend::u32 sm_block_enc[256];
                        __shared__ descend::u32 sm_kcmax[1];
                        {

                            descend::u64 codeword;
                            descend::u32 codewordlen;
                            descend::u32 kc;
                            descend::u32 startbit;

                            {
                                sm_cw[threadIdx.x] = p1[threadIdx.x];
                                sm_cwl[threadIdx.x] = p2[threadIdx.x];
                            }
                            __syncthreads(); // manually inserted!
                            {
                                {
                                    codeword = ((descend::u64) 0);
                                    codewordlen = 0u;
                                    const const descend::u32 tmp_d_item =
                                            p0[((blockIdx.x * 256) + threadIdx.x)];
                                    descend::u8 tmp_d_item_i = (descend::u8) ((tmp_d_item >> 24));
                                    descend::u32 tmpcw = sm_cw[tmp_d_item_i];
                                    descend::u32 tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64) (tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = (descend::u8) ((tmp_d_item >> 16));
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64) (tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = (descend::u8) ((tmp_d_item >> 8));
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64) (tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                    tmp_d_item_i = (descend::u8) (tmp_d_item);
                                    tmpcw = sm_cw[tmp_d_item_i];
                                    tmpcwl = sm_cwl[tmp_d_item_i];
                                    codeword = ((codeword << tmpcwl) | (descend::u64) (tmpcw));
                                    codewordlen = (codewordlen + tmpcwl);
                                }
                            }
                            descend::u32 tmp_shfl_res;
                            descend::u32 scan_codewordlen;
                            descend::u32 tmp_scan_block;

                            {

                                {
                                    scan_codewordlen = codewordlen;
                                    tmp_shfl_res =
                                            descend::shfl_up<descend::u32>(scan_codewordlen, 1);
                                }
                                if ((descend::warp().thread_rank() < 1)) {

                                } else {

                                    { scan_codewordlen = (scan_codewordlen + tmp_shfl_res); }
                                }
                                descend::warp().sync();

                                {
                                    tmp_shfl_res =
                                            descend::shfl_up<descend::u32>(scan_codewordlen, 2);
                                }
                                if ((descend::warp().thread_rank() < 2)) {

                                } else {

                                    { scan_codewordlen = (scan_codewordlen + tmp_shfl_res); }
                                }
                                descend::warp().sync();

                                {
                                    tmp_shfl_res =
                                            descend::shfl_up<descend::u32>(scan_codewordlen, 4);
                                }
                                if ((descend::warp().thread_rank() < 4)) {

                                } else {

                                    { scan_codewordlen = (scan_codewordlen + tmp_shfl_res); }
                                }
                                descend::warp().sync();

                                {
                                    tmp_shfl_res =
                                            descend::shfl_up<descend::u32>(scan_codewordlen, 8);
                                }
                                if ((descend::warp().thread_rank() < 8)) {

                                } else {

                                    { scan_codewordlen = (scan_codewordlen + tmp_shfl_res); }
                                }
                                descend::warp().sync();

                                {
                                    tmp_shfl_res =
                                            descend::shfl_up<descend::u32>(scan_codewordlen, 16);
                                }
                                if ((descend::warp().thread_rank() < 16)) {

                                } else {

                                    { scan_codewordlen = (scan_codewordlen + tmp_shfl_res); }
                                }
                                descend::warp().sync();
                            }
                            {

                                if ((descend::warp().meta_group_rank() < 7)) {

                                    {
                                        if ((descend::warp().thread_rank() < 31)) {

                                        } else {

                                            {
                                                sm_scan_arr[(descend::warp().meta_group_rank() - 0)] =
                                                        scan_codewordlen;
                                            }
                                        }
                                        descend::warp().sync();
                                    }
                                } else {
                                }
                                __syncthreads();
                            }
                            if ((descend::warp().meta_group_rank() < 1)) {

                                {
                                    if ((descend::warp().thread_rank() < 7)) {

                                        { tmp_scan_block = sm_scan_arr[(threadIdx.x - 0)]; }
                                    } else {

                                        { tmp_scan_block = 0u; }
                                    }
                                    descend::warp().sync();

                                    {
                                        tmp_shfl_res =
                                                descend::shfl_up<descend::u32>(tmp_scan_block, 1);
                                    }
                                    if ((descend::warp().thread_rank() < 1)) {

                                    } else {

                                        { tmp_scan_block = (tmp_scan_block + tmp_shfl_res); }
                                    }
                                    descend::warp().sync();

                                    {
                                        tmp_shfl_res =
                                                descend::shfl_up<descend::u32>(tmp_scan_block, 2);
                                    }
                                    if ((descend::warp().thread_rank() < 2)) {

                                    } else {

                                        { tmp_scan_block = (tmp_scan_block + tmp_shfl_res); }
                                    }
                                    descend::warp().sync();

                                    {
                                        tmp_shfl_res =
                                                descend::shfl_up<descend::u32>(tmp_scan_block, 4);
                                    }
                                    if ((descend::warp().thread_rank() < 4)) {

                                    } else {

                                        { tmp_scan_block = (tmp_scan_block + tmp_shfl_res); }
                                    }
                                    descend::warp().sync();
                                    {

                                        if ((descend::warp().thread_rank() < 7)) {

                                            { sm_scan_arr[(threadIdx.x - 0)] = tmp_scan_block; }
                                        } else {
                                        }
                                        descend::warp().sync();
                                    }
                                }
                            } else {
                            }
                            __syncthreads();
                            if ((descend::warp().meta_group_rank() < 1)) {

                            } else {

                                {

                                    {
                                        scan_codewordlen =
                                                (scan_codewordlen +
                                                 sm_scan_arr[(descend::warp().meta_group_rank() - 1)]);
                                    }
                                }
                            }
                            __syncthreads();

                            {

                                { scan_codewordlen = (scan_codewordlen - codewordlen); }
                            }

                            if ((threadIdx.x < 255)) {

                            } else {

                                {
                                    p4[((blockIdx.x * 1) + (threadIdx.x - 255))] =
                                            (scan_codewordlen + codewordlen);
                                    sm_kcmax[(threadIdx.x - 255)] =
                                            (scan_codewordlen + codewordlen) / 32u;
                                }
                            }
                            __syncthreads();
                            {
                                kc = (scan_codewordlen / 32);
                                startbit = (scan_codewordlen % 32);
                                descend::atomic_store(descend::atomic_ref<descend::u32>(
                                                              sm_block_enc[threadIdx.x]),
                                                      0u);
                            }
                            __syncthreads(); // manually inserted
                            {
                                descend::u32 wrbits;
                                if ((codewordlen > (32u - startbit))) {
                                    wrbits = (32u - startbit);
                                } else {
                                    wrbits = codewordlen;
                                }
                                descend::u32 tmpcw =
                                        (descend::u32) ((codeword >> (codewordlen - wrbits)));
                                descend::atomic_fetch_or(
                                        descend::atomic_ref<descend::u32>((*(&sm_block_enc[kc]))),
                                        (tmpcw << ((32u - startbit) - wrbits)));
                                codewordlen = (codewordlen - wrbits);
                                if ((codewordlen > 0u)) {
                                    if ((codewordlen > 32u)) {
                                        wrbits = 32u;
                                    } else {
                                        wrbits = codewordlen;
                                    }
                                    codewordlen = (codewordlen - wrbits);
                                    tmpcw = ((descend::u32) ((codeword >> codewordlen)) &
                                             ((1u << wrbits) - 1u));
                                    descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                                     (*(&sm_block_enc[(kc + 1)]))),
                                                             (tmpcw << (32u - wrbits)));
                                }
                                if ((codewordlen > 0u)) {
                                    tmpcw = (descend::u32) (
                                            (codeword & ((((descend::u64) 1) << codewordlen) -
                                                         ((descend::u64) 1))));
                                    descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                                     (*(&sm_block_enc[(kc + 2)]))),
                                                             (tmpcw << (32u - codewordlen)));
                                }

                                if ((descend::thread_id_x() <= sm_kcmax[0])) {
                                    p3[((blockIdx.x * 256) + threadIdx.x)] =
                                            descend::atomic_load(descend::atomic_ref<descend::u32>(
                                                    sm_block_enc[threadIdx.x]));
                                }
                            }
                        }
                    }
                },
                (&source_data), (&codewords), (&codewordlens), (&out_data), (&out_idx));
        descend::copy_to_host<descend::array<descend::u32, (64 * 256)>>((&out_data),
                                                                        h_out_data);
        descend::copy_to_host<descend::array<descend::u32, 64>>((&out_idx),
                                                                h_out_idx);
    }
}