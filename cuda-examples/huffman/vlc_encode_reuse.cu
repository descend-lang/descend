#include "hip/hip_runtime.h"
// =====================================================================================================================
//   code generated from ../examples/infer/huffman/vlc_encode_reuse.desc
// =====================================================================================================================

#include "descend.cuh"
/*
function declarations
*/
template <std::size_t gs>
__host__ auto vlc_encode(const descend::u32 *const h_source_data,
                         const descend::u32 *const h_codewords,
                         const descend::u32 *const h_codewordlens,
                         descend::u32 *const h_out,
                         descend::u32 *const h_out_idx) -> void;
template <std::size_t gs>
__global__ auto gpu_vlc_encode(const descend::u32 *const g_source_data,
                               const descend::u32 *const g_codewords,
                               const descend::u32 *const g_codewordlens,
                               descend::u32 *const g_out,
                               descend::u32 *const g_out_idx) -> void;
/*
function definitions
*/
template <std::size_t gs>
__host__ auto vlc_encode(const descend::u32 *const h_source_data,
                         const descend::u32 *const h_codewords,
                         const descend::u32 *const h_codewordlens,
                         descend::u32 *const h_out,
                         descend::u32 *const h_out_idx) -> void {
    auto gpu = descend::gpu_device(0);
    const auto g_source_data =
    descend::gpu_alloc_copy<descend::array<descend::u32, (gs * 256)>>(
            (&gpu), (&(*h_source_data)));
    const auto g_codewords =
    descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
            (&gpu), (&(*h_codewords)));
    const auto g_codewordlens =
    descend::gpu_alloc_copy<descend::array<descend::u32, 256>>(
            (&gpu), (&(*h_codewordlens)));
    auto g_out =
    descend::gpu_alloc_copy<descend::array<descend::u32, (gs * 256)>>(
            (&gpu), (&(*h_out)));
    auto g_out_idx = descend::gpu_alloc_copy<descend::array<descend::u32, gs>>(
            (&gpu), (&(*h_out_idx)));
    gpu_vlc_encode<gs>
    <<<dim3(gs, 1, 1), dim3(256, 1, 1),
    ((((0 + (4 * (1 * 256))) + (4 * (1 * 256))) + (4 * (1 * 256))) +
     (4 * (1 * 1)))>>>((&g_source_data), (&g_codewords), (&g_codewordlens),
                       (&g_out), (&g_out_idx));
    descend::copy_to_host<descend::array<descend::u32, (gs * 256)>>((&g_out),
            h_out);
    descend::copy_to_host<descend::array<descend::u32, gs>>((&g_out_idx),
            h_out_idx);
}

template <std::size_t gs>
__global__ auto gpu_vlc_encode(const descend::u32 *const g_source_data,
                               const descend::u32 *const g_codewords,
                               const descend::u32 *const g_codewordlens,
                               descend::u32 *const g_out,
                               descend::u32 *const g_out_idx) -> void {
    extern __shared__ descend::byte $buffer[];
    descend::u32 *const s_codewords = (descend::u32 *)((&$buffer[0]));
    descend::u32 *const s_codewordlens =
            (descend::u32 *)((&s_codewords[(1 * 256)]));
    descend::u32 *const s_scan_and_block_out =
            (descend::u32 *)((&s_codewordlens[(1 * 256)]));
    descend::u32 *const s_last_index_to_copy =
            (descend::u32 *)((&s_scan_and_block_out[(1 * 256)]));

    {

        auto l_thread_out = 0ull;
        auto l_thread_out_len = 0u;
        auto l_thread_start_value = 0u;
        auto l_thread_start_bit = 0u;
        {

            {
                const auto g_codeword_item = (&g_codewords[(threadIdx.x - 0)]);
                const auto g_codewordlen_item = (&g_codewordlens[(threadIdx.x - 0)]);
                const auto s_codeword_item = (&(&(*s_codewords))[(threadIdx.x - 0)]);
                const auto s_codewordlen_item =
                        (&(&(*s_codewordlens))[(threadIdx.x - 0)]);
                (*s_codeword_item) = (*g_codeword_item);
                (*s_codewordlen_item) = (*g_codewordlen_item);
            }
        }

        __syncthreads();
        {
            auto s_scan_ref = (&(*s_scan_and_block_out));

            {
                const auto g_source_data_item =
                        (&g_source_data[(((blockIdx.x - 0) * 256) + (threadIdx.x - 0))]);
                const auto s_scan_item = (&(&(*s_scan_ref))[(threadIdx.x - 0)]);
                const auto tmp_source_data_item = (*g_source_data_item);
                auto tmp_source_data_item_i = ((descend::u8)0);
                auto tmp_cw = 0u;
                auto tmp_cwl = 0u;
                for (std::size_t i = 0; (i < 4); i = (i + 1u)) {
                    tmp_source_data_item_i = (descend::u8)(
                            (tmp_source_data_item >> (8 * (3 - (descend::i32)(i)))));
                    tmp_cw = s_codewords[tmp_source_data_item_i];
                    tmp_cwl = s_codewordlens[tmp_source_data_item_i];
                    l_thread_out = ((l_thread_out << tmp_cwl) | (descend::u64)(tmp_cw));
                    l_thread_out_len = (l_thread_out_len + tmp_cwl);
                }

                (*s_scan_item) = l_thread_out_len;
            }
        }

        {
            auto s_scan_ref = (&(*s_scan_and_block_out));
            for (std::size_t d = 128; (d > 0u); d = (d >> 1)) {
                __syncthreads();
                if (((threadIdx.x - 0) < d)) {
                    {
                        (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                          ((256 / d) - 1))] =
                                ((&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((256 / d) - 1))] +
                                 (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((128 / d) - 1))]);
                    }
                } else {
                }
            }
        }

        {

            if (((threadIdx.x - 0) < 1)) {
                {
                    const auto last =
                            (&(&(*s_scan_and_block_out))[((threadIdx.x - 0) + (256 - 1))]);
                    (*last) = 0u;
                }
            } else {
            }
        }

        {
            auto s_scan_ref = (&(*s_scan_and_block_out));
            for (std::size_t d = 1; (d <= 128); d = (d * 2u)) {
                __syncthreads();
                if (((threadIdx.x - 0) < d)) {
                    {
                        const auto t = (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                         ((128 / d) - 1))];
                        (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                          ((128 / d) - 1))] =
                                (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                  ((256 / d) - 1))];
                        (&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                          ((256 / d) - 1))] =
                                ((&(*s_scan_ref))[(((threadIdx.x - 0) * (256 / d)) +
                                                   ((256 / d) - 1))] +
                                 t);
                    }
                } else {
                }
            }
        }

        __syncthreads();
        {

            if (((threadIdx.x - 0) < 255)) {
            } else {
                {
                    const auto g_out_idx_block_item =
                            (&g_out_idx[(((blockIdx.x - 0) * 1) + (threadIdx.x - 255))]);
                    const auto s_last_index_to_copy_item =
                            (&(&(*s_last_index_to_copy))[(threadIdx.x - 255)]);
                    (*g_out_idx_block_item) =
                            (s_scan_and_block_out[255] + l_thread_out_len);
                    (*s_last_index_to_copy_item) =
                            ((s_scan_and_block_out[255] + l_thread_out_len) / 32u);
                }
            }
        }

        {
            {
                const auto s_scan_and_block_out_item =
                        (&(&(*s_scan_and_block_out))[(threadIdx.x - 0)]);
                l_thread_start_value = ((*s_scan_and_block_out_item) / 32u);
                l_thread_start_bit = ((*s_scan_and_block_out_item) % 32u);
                (*s_scan_and_block_out_item) = 0u;
            }
        }

        __syncthreads();
        {
            auto s_block_out = (&(*s_scan_and_block_out));
            {
                descend::u32 wrbits;
                if ((l_thread_out_len > (32u - l_thread_start_bit))) {
                    wrbits = (32u - l_thread_start_bit);
                } else {
                    wrbits = l_thread_out_len;
                }

                auto tmpcw =
                        (descend::u32)((l_thread_out >> (l_thread_out_len - wrbits)));
                descend::atomic_fetch_or(
                        descend::atomic_ref<descend::u32>(
                                s_block_out[l_thread_start_value]),
                        (tmpcw << ((32u - l_thread_start_bit) - wrbits)));
                l_thread_out_len = (l_thread_out_len - wrbits);
                if ((l_thread_out_len > 0u)) {
                    if ((l_thread_out_len > 32u)) {
                        wrbits = 32u;
                    } else {
                        wrbits = l_thread_out_len;
                    }

                    tmpcw =
                            ((descend::u32)((l_thread_out >> (l_thread_out_len - wrbits))) &
                             ((1u << wrbits) - 1u));
                    descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                     s_block_out[(l_thread_start_value + 1)]),
                                             (tmpcw << (32u - wrbits)));
                    l_thread_out_len = (l_thread_out_len - wrbits);
                }

                if ((l_thread_out_len > 0u)) {
                    tmpcw = (descend::u32)(
                            (l_thread_out & ((1ull << l_thread_out_len) - 1ull)));
                    descend::atomic_fetch_or(descend::atomic_ref<descend::u32>(
                                                     s_block_out[(l_thread_start_value + 2)]),
                                             (tmpcw << (32u - l_thread_out_len)));
                }
            }
        }

        __syncthreads();
        {
            {
                const auto g_out_item =
                        (&g_out[(((blockIdx.x - 0) * 256) + (threadIdx.x - 0))]);
                const auto s_block_out_item =
                        (&(&(*s_scan_and_block_out))[(threadIdx.x - 0)]);
                if ((threadIdx.x <= s_last_index_to_copy[0])) {
                    (*g_out_item) = (*s_block_out_item);
                }
            }
        }
    }
}